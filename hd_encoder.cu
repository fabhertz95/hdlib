#include "hip/hip_runtime.h"
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

extern "C" {
#include "hd_encoder.h"
}

#define MAX_NUM_ITEMS 32

// TODO: Stuff to optimize
// * Copy one part of x to device, then compute it and copy the next part at the same time.
//   Use different streams for each part of the input, and use cudaMemcopyAsync.
// * make clip also on the gpu, using the data from before, and don't copy the ngramm_sum_buffer over.
template<int NGRAMM>
__global__ void hd_encoder_kernel(
    const int n_blk,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    const int n_items,
    const feature_t * __restrict__ x,
    const int n_x
)
{
    // compute the index of the block on which we must work
    int blk = blockIdx.x * blockDim.x + threadIdx.x;

    // exit if blk is outside of the range
    if (blk >= n_blk) {
        return;
    }

    // prepare local memory
    block_t l_ngramm_buffer[NGRAMM];
    uint32_t l_ngramm_sum_buffer[sizeof(uint32_t) * 8];
    block_t l_item_lookup[MAX_NUM_ITEMS];

    // reset ngramm_sum_buffer
    memset(l_ngramm_sum_buffer, 0, sizeof(l_ngramm_sum_buffer[0]) * 8 * sizeof(uint32_t));
    memset(l_ngramm_buffer, 0, sizeof(l_ngramm_buffer[0]) * NGRAMM);

    // load item_lookup
    uint32_t s_i; // iterator
    for (s_i = 0; s_i < n_items; s_i++) {
        l_item_lookup[s_i] = item_lookup[s_i * n_blk + blk];
    }

    // loop over every single feature
    int feat_idx;
    for (feat_idx = 0; feat_idx < n_x; feat_idx++) {
        // get position of the item in the lookup table
        feature_t item_lookup_idx = x[feat_idx];
        // get the part of the item
        block_t item = l_item_lookup[item_lookup_idx];

        // Shift the parts in in the item_buffer and add the new one
        int i;
        for (i = NGRAMM - 1; i >= 1; i--) {
            block_t previous = l_ngramm_buffer[i-1];
            l_ngramm_buffer[i] = (previous << 1) | (previous >> 31);
        }
        // set the new value
        l_ngramm_buffer[0] = item;

        // compute the encoded ngramm
        block_t tmp_ngramm_buffer = item;
        for (i = 1; i < NGRAMM; i++) {
            tmp_ngramm_buffer ^= l_ngramm_buffer[i];
        }

        // add to sum buffer
        if (feat_idx >= NGRAMM - 1) {
            int j;
            for (j = 0; j < sizeof(block_t) * 8; j++) {
                l_ngramm_sum_buffer[j] += (tmp_ngramm_buffer >> j) & 1;
            }
        }
    }

    // copy values back to ngramm_sum_buffer
    for (s_i = 0; s_i < sizeof(uint32_t) * 8; s_i++) {
        ngramm_sum_buffer[s_i * n_blk + blk] = l_ngramm_sum_buffer[s_i];
    }
}

// Wrapper function to call the kernel. Input data (x) must already be copied to the device.
// if stream is NULL, then the default stream is used.
extern "C" void hd_encoder_call_kernel(
    struct hd_encoder_t * const state,
    const feature_t * d_x,
    const int n_x,
    hipStream_t stream = NULL
)
{
    // compute the number of blocks used
    int num_blocks = (state->n_blk + NUM_THREADS_IN_BLOCK - 1) / NUM_THREADS_IN_BLOCK;

    switch(state->ngramm) {
        case 2:
            hd_encoder_kernel<2><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        case 3:
            hd_encoder_kernel<3><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        case 4:
            hd_encoder_kernel<4><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        case 5:
            hd_encoder_kernel<5><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        case 6:
            hd_encoder_kernel<6><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        case 7:
            hd_encoder_kernel<7><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        case 8:
            hd_encoder_kernel<8><<<num_blocks, NUM_THREADS_IN_BLOCK, 0, stream>>>(
                state->n_blk,
                state->device.ngramm_sum_buffer,
                state->device.item_lookup,
                state->n_items,
                d_x, n_x);
            break;
        default:
            printf("Error! ngramm must be between 2 and 8, but it was %d\n", state->ngramm);
    }
}

extern "C" void hd_encoder_setup_device(struct hd_encoder_t * const state) {
    // allocate memory
    hipMalloc(&(state->device.item_lookup), state->n_items * state->n_blk * sizeof(block_t));
    hipMalloc(&(state->device.ngramm_sum_buffer), state->n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
    hipMalloc(&(state->device.ngramm_buffer), state->ngramm * state->n_blk * sizeof(block_t));

    // copy LUT to device
    hipMemcpy(
        state->device.item_lookup,
        state->item_lookup,
        state->n_items * state->n_blk * sizeof(block_t),
        hipMemcpyHostToDevice
    );
}

extern "C" void hd_encoder_free(struct hd_encoder_t * const state) {
    hipFree(state->device.item_lookup);
    hipFree(state->device.ngramm_sum_buffer);
    hipFree(state->device.ngramm_buffer);

    free(state->ngramm_buffer);
    free(state->ngramm_sum_buffer);
    free(state->item_buffer);
    free(state->item_lookup);

    hipDeviceReset();
}

extern "C" void hd_encoder_encode (
    struct hd_encoder_t * const state,
    const feature_t * const x,
    const int n_x
)
{
    const int n_blk = state->n_blk;

    // reset the sum count
    state->ngramm_sum_count = 0;

    // allocate input data memory on the device
    feature_t * d_x;
    hipMalloc(&d_x, n_x * sizeof(feature_t));
    // copy the input data
    hipMemcpy(d_x, x, n_x * sizeof(feature_t), hipMemcpyHostToDevice);

    // call the kernel
    hd_encoder_call_kernel(state, d_x, n_x);

    hipDeviceSynchronize();

    // copy the output (ngramm_sum_buffer) back from the device
    hipMemcpy(
        state->ngramm_sum_buffer,
        state->device.ngramm_sum_buffer,
        n_blk * sizeof(block_t) * 8 * sizeof(uint32_t),
        hipMemcpyDeviceToHost
    );

    // free input memory
    hipFree(d_x);

    // set the ngramm_sum_count
    state->ngramm_sum_count += n_x - (state->ngramm - 1);
}

void clip(
    const uint32_t * const in,
    const int n_in,
    const int count,
    block_t * const out
)
{
    int threshold = count / 2;

    memset(out, 0, (n_in + sizeof(block_t) * 8 - 1) / (sizeof(block_t) * 8));

    // we ignore the randomization here...

    int n_blk = n_in / 32;
    int blk;
    for (blk = 0; blk < n_blk; blk++) {
        int s_i;
        for (s_i = 0; s_i < 32; s_i++) {
            out[blk] <<= 1;
            out[blk] += ((uint32_t)(threshold - in[s_i * n_blk + blk])) >> 31;
        }
    }

}

void hd_encoder_clip(
    struct hd_encoder_t * const state
)
{
    clip(
        state->ngramm_sum_buffer,
        sizeof(block_t) * 8 * state->n_blk,
        state->ngramm_sum_count,
        state->ngramm_buffer
    );
}