#include "hip/hip_runtime.h"
#include <string.h>
#include <stdint.h>
#include <stdlib.h>

extern "C" {
#include "hd_encoder.h"
}

// rand() generates a random number between 0 and RAND_MAX, which is
// guaranteed to be no less than 32767 on any standard implementation.
#if (RAND_MAX >= (1u << 32) - 1u)
#define RAND_BYTES 4
#elif (RAND_MAX >= (1u << 16) - 1u)
#define RAND_BYTES 2
#elif (RAND_MAX >= (1u << 8) - 1u)
#define RAND_BYTES 1
#endif

// number of threads per block in the grid
#define NUM_THREADS_IN_BLOCK 128

#define MAX_NUM_ITEMS 32
#define MAX_NGRAMM 8

// TODO: Stuff to optimize
// * Copy one part of x to device, then compute it and copy the next part at the same time.
//   Use different streams for each part of the input, and use cudaMemcopyAsync.
// * make clip also on the gpu, using the data from before, and don't copy the ngramm_sum_buffer over.
__global__ void hd_encoder_kernel(
    const int n_blk,
    const int ngramm,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    const int n_items,
    const feature_t * __restrict__ x,
    const int n_x
)
{
    // compute the index of the block on which we must work
    int blk = blockIdx.x*blockDim.x + threadIdx.x;

    // exit if blk is outside of the range
    if (blk >= n_blk) {
        return;
    }

    // prepare local memory
    block_t l_ngramm_buffer[MAX_NGRAMM];
    uint32_t l_ngramm_sum_buffer[sizeof(uint32_t) * 8];
    block_t l_item_lookup[MAX_NUM_ITEMS];
    
    // reset ngramm_sum_buffer
    memset(l_ngramm_sum_buffer, 0, sizeof(l_ngramm_sum_buffer[0]) * 8 * sizeof(uint32_t));
    memset(l_ngramm_buffer, 0, sizeof(l_ngramm_buffer[0]) * ngramm);

    // load item_lookup
    uint32_t s_i; // iterator
    for (s_i = 0; s_i < n_items; s_i++) {
        l_item_lookup[s_i] = item_lookup[s_i * n_blk + blk];
    }

    // loop over every single feature
    int feat_idx;
    for (feat_idx = 0; feat_idx < n_x; feat_idx++) {
        // get position of the item in the lookup table
        feature_t item_lookup_idx = x[feat_idx];
        // get the part of the item
        block_t item = l_item_lookup[item_lookup_idx];

        // Shift the parts in in the item_buffer and add the new one
        int i;
        for (i = ngramm - 1; i >= 1; i--) {
            block_t previous = l_ngramm_buffer[i-1];
            l_ngramm_buffer[i] = (previous << 1) | (previous >> 31);
        }
        // set the new value
        l_ngramm_buffer[0] = item;

        // compute the encoded ngramm
        block_t tmp_ngramm_buffer = item;
        for (i = 1; i < ngramm; i++) {
            tmp_ngramm_buffer ^= l_ngramm_buffer[i];
        }

        // add to sum buffer
        if (feat_idx >= ngramm - 1) {
            int j;
            for (j = 0; j < sizeof(block_t) * 8; j++) {
                l_ngramm_sum_buffer[j] += (tmp_ngramm_buffer >> j) & 1;
            }
        }
    }

    // copy values back to ngramm_sum_buffer
    // TODO reorder sum buffer such that we can just use memcopy
    for (s_i = 0; s_i < sizeof(uint32_t) * 8; s_i++) {
        ngramm_sum_buffer[blk * sizeof(uint32_t) * 8 + s_i] = l_ngramm_sum_buffer[s_i];
    }
}

extern "C" void hd_encoder_init(
    struct hd_encoder_t * const state,
    const int n_blk,
    const int ngramm,
    const int n_items
)
{
    state->n_blk = n_blk;
    state->ngramm = ngramm;
    state->n_items = n_items;
    state->ngramm_buffer = (block_t*)malloc(n_blk * sizeof(block_t));
    state->ngramm_sum_buffer = (uint32_t*)malloc(n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
    state->item_buffer = (block_t*)malloc(ngramm * n_blk * sizeof(block_t));
    state->item_buffer_head = 0;
    state->item_lookup = (block_t*)malloc(n_items * n_blk * sizeof(block_t));

    // initialise HD vector lookup table with uniformly distributed 0s and 1s
    int i;
    for (i = 0; i < n_items * n_blk; ++i)
    {
        state->item_lookup[i] = 0;

        int j;
        for (j = 0; j < sizeof(state->item_lookup[0]) / RAND_BYTES; j++)
        {
            state->item_lookup[i] <<= 8 * RAND_BYTES;
            state->item_lookup[i] += rand() & ((1u << 8 * RAND_BYTES) - 1u);
        }
    }
}

extern "C" void hd_encoder_setup_device(struct hd_encoder_t * const state) {
    // allocate memory
    hipMalloc(&(state->device.item_lookup), state->n_items * state->n_blk * sizeof(block_t));
    hipMalloc(&(state->device.ngramm_sum_buffer), state->n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
    hipMalloc(&(state->device.ngramm_buffer), state->ngramm * state->n_blk * sizeof(block_t));

    // copy LUT to device
    hipMemcpy(
        state->device.item_lookup,
        state->item_lookup,
        state->n_items * state->n_blk * sizeof(block_t),
        hipMemcpyHostToDevice
    );
}

extern "C" void hd_encoder_free(struct hd_encoder_t * const state) {
    hipFree(state->device.item_lookup);
    hipFree(state->device.ngramm_sum_buffer);
    hipFree(state->device.ngramm_buffer);

    free(state->ngramm_buffer);
    free(state->ngramm_sum_buffer);
    free(state->item_buffer);
    free(state->item_lookup);
}

extern "C" void hd_encoder_encode (
    struct hd_encoder_t * const state,
    const feature_t * const x,
    const int n_x
)
{
    const int n_blk = state->n_blk;
    const int ngramm = state->ngramm;
    const int n_items = state->n_items;

    // reset the sum count
    state->ngramm_sum_count = 0;

    // allocate input data memory on the device
    feature_t * d_x;
    hipMalloc(&d_x, n_x * sizeof(feature_t));
    // copy the input data
    hipMemcpy(d_x, x, n_x * sizeof(feature_t), hipMemcpyHostToDevice);

    // call the kernel
    int num_blocks = (n_blk + NUM_THREADS_IN_BLOCK - 1) / NUM_THREADS_IN_BLOCK;

    hd_encoder_kernel<<<num_blocks, NUM_THREADS_IN_BLOCK>>>(
        n_blk,
        ngramm,
        state->device.ngramm_sum_buffer,
        state->device.item_lookup,
        n_items,
        d_x,
        n_x
    );

    // copy the output (ngramm_sum_buffer) back from the device
    hipMemcpy(
        state->ngramm_sum_buffer,
        state->device.ngramm_sum_buffer,
        n_blk * sizeof(block_t) * 8 * sizeof(uint32_t),
        hipMemcpyDeviceToHost
    );

    // free input memory
    hipFree(d_x);

    // set the ngramm_sum_count
    state->ngramm_sum_count += n_x - (state->ngramm - 1);
}

extern "C" void hd_encoder_clip(
    const uint32_t * const in,
    const int n_in,
    const int count,
    block_t * const out
)
{
    int threshold = count / 2;

    memset(out, 0, (n_in + sizeof(block_t) * 8 - 1) / (sizeof(block_t) * 8));

    // add a random vector to break ties if case an even number of elements were summed
    if (count % 2 == 0)
    {
        // TODO: can we reuse randomness? e.g. have a fixed length of say 32 bytes
        uint32_t random_vector[(n_in + 31) / 32];
        int i;
        for (i = 0; i > sizeof(random_vector) / sizeof(random_vector[0]); i++)
        {
            random_vector[i] = 0;
            int j;
            for (j = 0; j < RAND_BYTES; j++)
            {
                random_vector[i] <<= 8 * RAND_BYTES;
                random_vector[i] += rand() & ((1u << 8 * RAND_BYTES) - 1u);
            }
        }

        for (i = 0; i < n_in; i++)
        {
            int in_with_rand = in[i] + (random_vector[i / 32] & 1);
            random_vector[i / 32] >>= 1;
            out[i / 32] <<= 1;
            // set to 1 if above threshold and 0 otherwise
            out[i / 32] += ((uint32_t)(threshold - in_with_rand)) >> 31;
        }
    }
    else
    {
        int i;
        for (i = 0; i < n_in; i++)
        {
            out[i / 32] <<= 1;
            out[i / 32] += ((uint32_t)(threshold - in[i])) >> 31;
        }
    }

}
