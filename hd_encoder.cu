#include "hip/hip_runtime.h"
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

extern "C" {
#include "hd_encoder.h"
}

// rand() generates a random number between 0 and RAND_MAX, which is
// guaranteed to be no less than 32767 on any standard implementation.
#if (RAND_MAX >= (1u << 32) - 1u)
#define RAND_BYTES 4
#elif (RAND_MAX >= (1u << 16) - 1u)
#define RAND_BYTES 2
#elif (RAND_MAX >= (1u << 8) - 1u)
#define RAND_BYTES 1
#endif

// number of threads per block in the grid
#define NUM_THREADS_IN_BLOCK 128

__global__ void hd_encoder_kernel(
    const int n_blk,
    const int ngramm,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    block_t * __restrict__ item_buffer,
    const feature_t * __restrict__ x,
    const int n_x
)
{
    // compute the index of the block on which we must work
    int blk = blockIdx.x*blockDim.x + threadIdx.x;

    // exit if blk is outside of the range
    if (blk >= n_blk) {
        return;
    }

    // loop over every single feature
    int feat_idx;
    for (feat_idx = 0; feat_idx < n_x; feat_idx++) {
        // get position of the item in the lookup table
        feature_t item_lookup_idx = x[feat_idx];
        // get the part of the item
        block_t item = *(item_lookup + item_lookup_idx * n_blk + blk);

        // Shift the parts in in the item_buffer and add the new one
        int i;
        for (i = ngramm - 1; i >= 1; i--) {
            block_t previous = item_buffer[(i-1) * n_blk + blk];
            item_buffer[i * n_blk + blk] = (previous << 1) | (previous >> 31);
        }
        // set the new value
        item_buffer[blk] = item;

        // compute the encoded ngramm
        block_t tmp_ngramm_buffer = item;
        for (i = 1; i < ngramm; i++) {
            tmp_ngramm_buffer ^= item_buffer[i * n_blk + blk];
        }

        // add to sum buffer
        if (feat_idx >= ngramm - 1) {
            uint32_t * ngramm_sum_buffer_iter = ngramm_sum_buffer + blk * sizeof(block_t) * 8;
            int j;
            for (j = 0; j < sizeof(block_t) * 8; j++) {
                *ngramm_sum_buffer_iter++ += ((tmp_ngramm_buffer) >> j) & 1;
            }
        }
    }
}

extern "C" void hd_encoder_init(
    struct hd_encoder_t * const state,
    const int n_blk,
    const int ngramm,
    const int n_items
)
{
    state->n_blk = n_blk;
    state->ngramm = ngramm;
    state->n_items = n_items;
    state->ngramm_buffer = (block_t*)malloc(n_blk * sizeof(block_t));
    state->ngramm_sum_buffer = (uint32_t*)malloc(n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
    state->item_buffer = (block_t*)malloc(ngramm * n_blk * sizeof(block_t));
    state->item_buffer_head = 0;
    state->item_lookup = (block_t*)malloc(n_items * n_blk * sizeof(block_t));

    // initialise HD vector lookup table with uniformly distributed 0s and 1s
    int i;
    for (i = 0; i < n_items * n_blk; ++i)
    {
        state->item_lookup[i] = 0;

        int j;
        for (j = 0; j < sizeof(state->item_lookup[0]) / RAND_BYTES; j++)
        {
            state->item_lookup[i] <<= 8 * RAND_BYTES;
            state->item_lookup[i] += rand() & ((1u << 8 * RAND_BYTES) - 1u);
        }
    }
}

extern "C" void hd_encoder_encode (
    struct hd_encoder_t * const state,
    const feature_t * const x,
    const int n_x
)
{
    const int n_blk = state->n_blk;
    const int ngramm = state->ngramm;
    const int n_items = state->n_items;

    // allocate memory on the device
    uint32_t * d_ngramm_sum_buffer;
    block_t * d_item_buffer;
    feature_t * d_x;
    printf("allocate ngramm_sum_buffer: %d\n", n_blk * sizeof(block_t) * 8);
    // TODO hangs here
    hipMalloc(&d_ngramm_sum_buffer, n_blk * sizeof(block_t) * 8);
    printf("allocate item_buffer: %d\n", ngramm * n_blk * sizeof(block_t));
    hipMalloc(&d_item_buffer, ngramm * n_blk * sizeof(block_t));
    printf("allocate x: %d\n", n_x * sizeof(feature_t));
    hipMalloc(&d_x, n_x * sizeof(feature_t));

    // TODO allocate and copy these values in some init function, because they will remain constant for all samples
    block_t * d_item_lookup;
    printf("allocate item_lookup: %d\n", n_items * n_blk * sizeof(block_t));
    hipMalloc(&d_item_lookup, n_items * n_blk * sizeof(block_t));

    // reset sum buffer and item buffer
    hipMemset(d_ngramm_sum_buffer, 0, n_blk * sizeof(block_t) * 8);
    hipMemset(d_item_buffer, 0, ngramm * n_blk * sizeof(block_t));

    // copy the item lookup
    // TODO allocate and copy these values in some init function, because they will remain constant for all samples
    hipMemcpy(d_item_lookup, state->item_lookup, n_items * n_blk * sizeof(block_t), hipMemcpyHostToDevice);
    // copy the input data
    hipMemcpy(d_x, x, n_x * sizeof(feature_t), hipMemcpyHostToDevice);

    // call the kernel
    printf("call kernel\n");
    int num_blocks = (n_blk + NUM_THREADS_IN_BLOCK - 1) / NUM_THREADS_IN_BLOCK;
    hd_encoder_kernel<<<num_blocks, NUM_THREADS_IN_BLOCK>>>(
        n_blk,
        ngramm,
        d_ngramm_sum_buffer,
        d_item_lookup,
        d_item_buffer,
        d_x,
        n_x
    );

    // copy the output (ngramm_sum_buffer) back from the device
    hipMemcpy(state->ngramm_sum_buffer, d_ngramm_sum_buffer, n_blk * sizeof(block_t) * 8, hipMemcpyDeviceToHost);

    // free all memory
    hipFree(d_ngramm_sum_buffer);
    hipFree(d_item_buffer);
    hipFree(d_item_lookup);
    hipFree(d_x);

    // set the ngramm_sum_count
    state->ngramm_sum_count += n_x - (state->ngramm - 1);
}

extern "C" void hd_encoder_clip(
    const uint32_t * const in,
    const int n_in,
    const int count,
    block_t * const out
)
{
    int threshold = count / 2;

    memset(out, 0, (n_in + sizeof(block_t) * 8 - 1) / (sizeof(block_t) * 8));

    // add a random vector to break ties if case an even number of elements were summed
    if (count % 2 == 0)
    {
        // TODO: can we reuse randomness? e.g. have a fixed length of say 32 bytes
        uint32_t random_vector[(n_in + 31) / 32];
        int i;
        for (i = 0; i > sizeof(random_vector) / sizeof(random_vector[0]); i++)
        {
            random_vector[i] = 0;
            int j;
            for (j = 0; j < RAND_BYTES; j++)
            {
                random_vector[i] <<= 8 * RAND_BYTES;
                random_vector[i] += rand() & ((1u << 8 * RAND_BYTES) - 1u);
            }
        }

        for (i = 0; i < n_in; i++)
        {
            int in_with_rand = in[i] + (random_vector[i / 32] & 1);
            random_vector[i / 32] >>= 1;
            out[i / 32] <<= 1;
            // set to 1 if above threshold and 0 otherwise
            out[i / 32] += ((uint32_t)(threshold - in_with_rand)) >> 31;
        }
    }
    else
    {
        int i;
        for (i = 0; i < n_in; i++)
        {
            out[i / 32] <<= 1;
            out[i / 32] += ((uint32_t)(threshold - in[i])) >> 31;
        }
    }

}
