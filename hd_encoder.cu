#include "hip/hip_runtime.h"
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

extern "C" {
#include "hd_encoder.h"
}

// number of threads per block in the grid
#define NUM_THREADS_IN_BLOCK 128
#define MAX_NUM_INPUT_CHUNKS 4

#define NUM_HD_BLOCKS_IN_BLOCK (NUM_THREADS_IN_BLOCK / MAX_NUM_INPUT_CHUNKS)

#define MAX_NUM_ITEMS 32

// TODO: Stuff to optimize
// * Copy one part of x to device, then compute it and copy the next part at the same time.
//   Use different streams for each part of the input, and use cudaMemcopyAsync.
// * make clip also on the gpu, using the data from before, and don't copy the ngramm_sum_buffer over.

// encode the whole input with a chunk of the HD vector (a single)
template<int NGRAMM, int NUM_INPUT_CHUNKS>
__global__ void hd_encoder_kernel(
    const int n_blk,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    const int n_items,
    const feature_t * __restrict__ x,
    const int n_x
)
{
    // setup shared memory
    extern __shared__ uint32_t s[];

    // compute the index of the block on which we must work
    int blk_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x_chunk_idx = blockIdx.y * blockDim.y + threadIdx.y;

    // exit if blk_idx is outside of the range
    if (blk_idx >= n_blk || x_chunk_idx >= NUM_INPUT_CHUNKS) {
        return;
    }

    // there are (NUM_INPUT_CHUNKS - 1) overlaps of (NGRAMM - 1),
    // but the number of XOR calculations only depends on
    // the number of outputs to the accummulation buffer

    // example: n_x = 9, ngramm = 3, NUM_INPUT_CHUNKS = 2
    //          xored_n_x := 7, x_chunk_len = 4 (3 for last chunk)
    //
    // o - load only, x - load and xor
    // chunk 0: ooxxxx---
    // chunk 1: ----ooxxx
    //            <--> chunk 0 length (pre-loaded data not included)
    const int xored_n_x = n_x - (NGRAMM - 1);
    int x_chunk_len = (xored_n_x + NUM_INPUT_CHUNKS - 1) / NUM_INPUT_CHUNKS;
    const int x_chunk_start = x_chunk_idx * x_chunk_len;
    if (x_chunk_start + x_chunk_len > n_x) {
        x_chunk_len = n_x - x_chunk_start;
    }

    int i; // iterator

    // local copies of the:
    // - HD feature vector chunk n-gramm buffer
    block_t l_item_buffer[NGRAMM];
    memset(l_item_buffer, 0, sizeof(l_item_buffer));

    // - encoded n-gramm summation chunk buffer
    uint32_t l_ngramm_sum_buffer[sizeof(block_t) * 8];
    memset(l_ngramm_sum_buffer, 0, sizeof(l_ngramm_sum_buffer));

    // - HD vector chunk lookup array
    // To load the lookup array, we should use every thread for fetching the data.
    // Split n_items into chunks, to load the lookup of a single block in parallel.
    int num_items_to_load = (n_items + NUM_INPUT_CHUNKS - 1) / NUM_INPUT_CHUNKS;
    int start_item_to_load = x_chunk_idx * num_items_to_load;
    if (start_item_to_load + num_items_to_load > n_items) {
        num_items_to_load = n_items - start_item_to_load;
    }
    block_t *  s_item_lookup = s;
    for (i = start_item_to_load; i < start_item_to_load + num_items_to_load; i++) {
        s_item_lookup[i * blockDim.x + threadIdx.x] = item_lookup[i * n_blk + blk_idx];
    }

    // sync threads if NUM_INPUT_CHUNKS is bigger than 1 (else, we do not have any dependeny)
    if (NUM_INPUT_CHUNKS > 1) {
        __syncthreads();
    }

    // loop over every single feature
    int x_chunk_iter;
    for (x_chunk_iter = 0; x_chunk_iter < NGRAMM - 1 + x_chunk_len; x_chunk_iter++) {
        // barrel shift each HD feature vector chunk as it gets a feature increment older
        int i;
        for (i = NGRAMM - 1; i >= 1; i--) {
            block_t previous = l_item_buffer[i-1];
            l_item_buffer[i] = (previous << 1) | (previous >> 31);
        }

        // populate new HD feature vector chunk
        feature_t item_lookup_idx = x[x_chunk_start + x_chunk_iter];
        block_t item = s_item_lookup[item_lookup_idx * blockDim.x + threadIdx.x];
        l_item_buffer[0] = item;

        // only pre-load the first (NGRAMM - 1) items
        if (x_chunk_iter >= NGRAMM - 1) {
            // compute the encoded n-gramm
            block_t tmp_ngramm_buffer = item;
            for (i = 1; i < NGRAMM; i++) {
                tmp_ngramm_buffer ^= l_item_buffer[i];
            }
    
            // unpack and accumulate the encoded n-gramm
            for (i = 0; i < sizeof(block_t) * 8; i++) {
                l_ngramm_sum_buffer[i] += (tmp_ngramm_buffer >> i) & 1;
            }
        }
    }

    // accumulating the results to the ngramm_sum_buffer creates a memory race condition
    // avoid this by means of linear reduction across threads
    // reduction can use shared memory
    // TODO implement a better reduction
    if (NUM_INPUT_CHUNKS > 1) {
        // make sure that all threads are done
        __syncthreads();
        // reuse shared memory for the reduction
        uint32_t * s_ngramm_sum_buffer = s;

        int curr_x_chunk = 0;
        // make the first chunk (overwrite values)
        if (curr_x_chunk == x_chunk_idx) {
            for (i = 0; i < sizeof(block_t) * 8; i++) {
                s_ngramm_sum_buffer[i * blockDim.x + threadIdx.x] = l_ngramm_sum_buffer[i];
            }
        }

        // copy all remaining chunks except the last one
        for (curr_x_chunk = 1; curr_x_chunk < (NUM_INPUT_CHUNKS - 1); curr_x_chunk++) {
            __syncthreads();
            if (curr_x_chunk == x_chunk_idx) {
                // copy values back to ngramm_sum_buffer
                for (i = 0; i < sizeof(block_t) * 8; i++) {
                    s_ngramm_sum_buffer[i * blockDim.x + threadIdx.x] += l_ngramm_sum_buffer[i];
                }
            }
        }

        // add the last one and copy into global (result) memory
        curr_x_chunk = NUM_INPUT_CHUNKS - 1;
        __syncthreads();
        if (curr_x_chunk == x_chunk_idx) {
            // copy values back to ngramm_sum_buffer
            for (i = 0; i < sizeof(block_t) * 8; i++) {
                ngramm_sum_buffer[i * n_blk + blk_idx] = l_ngramm_sum_buffer[i] + s_ngramm_sum_buffer[i * blockDim.x + threadIdx.x];
            }
        }
    } else {
        for (i = 0; i < sizeof(block_t) * 8; i++) {
            ngramm_sum_buffer[i * n_blk + blk_idx] = l_ngramm_sum_buffer[i];
        }
    }
}

// Wrapper function to call the kernel. Input data (x) must already be copied to the device.
// if stream is NULL, then the default stream is used.
extern "C" void hd_encoder_call_kernel(
    struct hd_encoder_t * const state,
    const feature_t * d_x,
    const int n_x,
    int use_input_chunks = 1,
    hipStream_t stream = NULL
)
{
    dim3 threads, grid;
    int smem_size;

    // conmpute the maximum of n_items and 32, to make sure we have enough space for item lookup and reduction
    int smem_parts;
    if (state->n_items > sizeof(block_t) * 8) {
        smem_parts = state->n_items;
    } else {
        smem_parts = sizeof(block_t) * 8;
    }

    if (use_input_chunks) {
        // Each grid block calculates a chunk of the HD vector
        // for the entire input. Withing the block, threads divide work
        // both along the HD vector and the input.
        threads.x = NUM_HD_BLOCKS_IN_BLOCK;
        threads.y = MAX_NUM_INPUT_CHUNKS;

        // compute the number of blocks used
        grid.x = (state->n_blk + NUM_HD_BLOCKS_IN_BLOCK - 1) / NUM_HD_BLOCKS_IN_BLOCK;

        smem_size = smem_parts * NUM_HD_BLOCKS_IN_BLOCK * sizeof(block_t);
    } else {
        threads.x = NUM_THREADS_IN_BLOCK;
        grid.x = (state->n_blk + NUM_THREADS_IN_BLOCK - 1) / NUM_THREADS_IN_BLOCK;
        smem_size = smem_parts * NUM_THREADS_IN_BLOCK * sizeof(block_t);
    }

    switch(state->ngramm) {
#define CALL_KERNEL_CASE(N) \
        case N: \
            if (use_input_chunks) { \
                hd_encoder_kernel<N, MAX_NUM_INPUT_CHUNKS><<<grid, threads, smem_size, stream>>>(    \
                    state->n_blk, \
                    state->device.ngramm_sum_buffer, \
                    state->device.item_lookup, \
                    state->n_items, \
                    d_x, n_x); \
            } else { \
                hd_encoder_kernel<N, 1><<<grid, threads, smem_size, stream>>>(    \
                    state->n_blk, \
                    state->device.ngramm_sum_buffer, \
                    state->device.item_lookup, \
                    state->n_items, \
                    d_x, n_x); \
            } \
            break;

        CALL_KERNEL_CASE(2)
        CALL_KERNEL_CASE(3)
        CALL_KERNEL_CASE(4)
        CALL_KERNEL_CASE(5)
        CALL_KERNEL_CASE(6)
        CALL_KERNEL_CASE(7)
        CALL_KERNEL_CASE(8)

        default:
            printf("Error! ngramm must be between 2 and 8, but it was %d\n", state->ngramm);
    }
}

extern "C" void hd_encoder_setup_device(struct hd_encoder_t * const state) {
    // allocate memory
    hipMalloc(&(state->device.item_lookup), state->n_items * state->n_blk * sizeof(block_t));
    hipMalloc(&(state->device.ngramm_sum_buffer), state->n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
    hipMalloc(&(state->device.ngramm_buffer), state->ngramm * state->n_blk * sizeof(block_t));

    // copy LUT to device
    hipMemcpy(
        state->device.item_lookup,
        state->item_lookup,
        state->n_items * state->n_blk * sizeof(block_t),
        hipMemcpyHostToDevice
    );
}

extern "C" void hd_encoder_free(struct hd_encoder_t * const state) {
    hipFree(state->device.item_lookup);
    hipFree(state->device.ngramm_sum_buffer);
    hipFree(state->device.ngramm_buffer);

    free(state->ngramm_buffer);
    free(state->ngramm_sum_buffer);
    free(state->item_buffer);
    free(state->item_lookup);

    hipDeviceReset();
}

extern "C" void hd_encoder_encode (
    struct hd_encoder_t * const state,
    const feature_t * const x,
    const int n_x
)
{
    const int n_blk = state->n_blk;

    // reset the sum count and buffer
    state->ngramm_sum_count = 0;
    hipMemset(
        state->device.ngramm_sum_buffer,
        0,
        n_blk * sizeof(block_t) * 8 * sizeof(uint32_t)
    );

    // allocate input data memory on the device
    feature_t * d_x;
    hipMalloc(&d_x, n_x * sizeof(feature_t));
    // copy the input data
    hipMemcpy(d_x, x, n_x * sizeof(feature_t), hipMemcpyHostToDevice);

    // call the kernel
    hd_encoder_call_kernel(state, d_x, n_x);

    hipDeviceSynchronize();

    // copy the output (ngramm_sum_buffer) back from the device
    hipMemcpy(
        state->ngramm_sum_buffer,
        state->device.ngramm_sum_buffer,
        n_blk * sizeof(block_t) * 8 * sizeof(uint32_t),
        hipMemcpyDeviceToHost
    );

    // free input memory
    hipFree(d_x);

    // set the ngramm_sum_count
    state->ngramm_sum_count += n_x - (state->ngramm - 1);
}

void clip(
    const uint32_t * const in,
    const int n_in,
    const int count,
    block_t * const out
)
{
    int threshold = count / 2;

    memset(out, 0, (n_in + sizeof(block_t) * 8 - 1) / (sizeof(block_t) * 8));

    // we ignore the randomization here...

    int n_blk = n_in / 32;
    int blk_idx;
    for (blk_idx = 0; blk_idx < n_blk; blk_idx++) {
        int i;
        for (i = 0; i < 32; i++) {
            out[blk_idx] <<= 1;
            out[blk_idx] += ((uint32_t)(threshold - in[i * n_blk + blk_idx])) >> 31;
        }
    }

}

void hd_encoder_clip(
    struct hd_encoder_t * const state
)
{
    clip(
        state->ngramm_sum_buffer,
        sizeof(block_t) * 8 * state->n_blk,
        state->ngramm_sum_count,
        state->ngramm_buffer
    );
}