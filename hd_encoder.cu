#include "hip/hip_runtime.h"
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

extern "C" {
#include "hd_encoder.h"
}

// number of threads per block in the grid
#define NUM_THREADS_IN_BLOCK 128
#define NUM_INPUT_CHUNKS 2

#define NUM_HD_BLOCKS_IN_BLOCK (NUM_THREADS_IN_BLOCK / NUM_INPUT_CHUNKS)

#define MAX_NUM_ITEMS 32

// TODO: Stuff to optimize
// * Copy one part of x to device, then compute it and copy the next part at the same time.
//   Use different streams for each part of the input, and use cudaMemcopyAsync.
// * make clip also on the gpu, using the data from before, and don't copy the ngramm_sum_buffer over.

// encode the whole input with a chunk of the HD vector (a single)
template<int NGRAMM>
__global__ void hd_encoder_kernel(
    const int n_blk,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    const int n_items,
    const feature_t * __restrict__ x,
    const int n_x
)
{
    // compute the index of the block on which we must work
    int blk_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x_chunk_idx = blockIdx.y * blockDim.y + threadIdx.y;

    // exit if blk_idx is outside of the range
    if (blk_idx >= n_blk || x_chunk_idx >= NUM_INPUT_CHUNKS) {
        return;
    }

    // there are (NUM_INPUT_CHUNKS - 1) overlaps of (NGRAMM - 1),
    // but the number of XOR calculations only depends on
    // the number of outputs to the accummulation buffer

    // example: n_x = 9, ngramm = 3, NUM_INPUT_CHUNKS = 2
    //          xored_n_x := 7, x_chunk_len = 4 (3 for last chunk)
    //
    // o - load only, x - load and xor
    // chunk 0: ooxxxx---
    // chunk 1: ----ooxxx
    //            <--> chunk 0 length (pre-loaded data not included)
    const int xored_n_x = n_x - (NGRAMM - 1);
    int x_chunk_len = (xored_n_x + NUM_INPUT_CHUNKS - 1) / NUM_INPUT_CHUNKS;
    const int x_chunk_start = x_chunk_idx * x_chunk_len;
    if (x_chunk_start + x_chunk_len > n_x) {
        x_chunk_len = n_x - x_chunk_start;
    }

    int i; // iterator

    // local copies of the:
    // - HD feature vector chunk n-gramm buffer
    block_t l_item_buffer[NGRAMM];
    memset(l_item_buffer, 0, sizeof(l_item_buffer));

    // - encoded n-gramm summation chunk buffer
    uint32_t l_ngramm_sum_buffer[sizeof(block_t) * 8];
    memset(l_ngramm_sum_buffer, 0, sizeof(l_ngramm_sum_buffer));

    // - HD vector chunk lookup array
    block_t l_item_lookup[MAX_NUM_ITEMS];
    for (i = 0; i < n_items; i++) {
        l_item_lookup[i] = item_lookup[i * n_blk + blk_idx];
    }

    // loop over every single feature
    int x_chunk_iter;
    for (x_chunk_iter = 0; x_chunk_iter < NGRAMM - 1 + x_chunk_len; x_chunk_iter++) {
        // barrel shift each HD feature vector chunk as it gets a feature increment older
        int i;
        for (i = NGRAMM - 1; i >= 1; i--) {
            block_t previous = l_item_buffer[i-1];
            l_item_buffer[i] = (previous << 1) | (previous >> 31);
        }

        // populate new HD feature vector chunk
        feature_t item_lookup_idx = x[x_chunk_start + x_chunk_iter];
        block_t item = l_item_lookup[item_lookup_idx];
        l_item_buffer[0] = item;

        // only pre-load the first (NGRAMM - 1) items
        if (x_chunk_iter >= NGRAMM - 1) {
            // compute the encoded n-gramm
            block_t tmp_ngramm_buffer = item;
            for (i = 1; i < NGRAMM; i++) {
                tmp_ngramm_buffer ^= l_item_buffer[i];
            }
    
            // unpack and accumulate the encoded n-gramm
            for (i = 0; i < sizeof(block_t) * 8; i++) {
                l_ngramm_sum_buffer[i] += (tmp_ngramm_buffer >> i) & 1;
            }
        }
    }

    // accumulating the results to the ngramm_sum_buffer creates a memory race condition
    // avoid this by means of linear reduction across threads
    // TODO implement a better reduction
    int curr_x_chunk;
    for (curr_x_chunk = 0; curr_x_chunk < NUM_INPUT_CHUNKS; curr_x_chunk++) {
        __syncthreads();
        if (curr_x_chunk == x_chunk_idx) {
            // copy values back to ngramm_sum_buffer
            for (i = 0; i < sizeof(block_t) * 8; i++) {
                ngramm_sum_buffer[i * n_blk + blk_idx] += l_ngramm_sum_buffer[i];
            }
        }
    }
}

// Wrapper function to call the kernel. Input data (x) must already be copied to the device.
// if stream is NULL, then the default stream is used.
extern "C" void hd_encoder_call_kernel(
    struct hd_encoder_t * const state,
    const feature_t * d_x,
    const int n_x,
    hipStream_t stream = NULL
)
{
    // Each grid block calculates a chunk of the HD vector
    // for the entire input. Withing the block, threads divide work
    // both along the HD vector and the input.
    dim3 threads(NUM_THREADS_IN_BLOCK / NUM_INPUT_CHUNKS, NUM_INPUT_CHUNKS);

    // compute the number of blocks used
    int num_blocks = (state->n_blk + NUM_HD_BLOCKS_IN_BLOCK - 1) / NUM_HD_BLOCKS_IN_BLOCK;

    dim3 grid(num_blocks);

    switch(state->ngramm) {
#define CALL_KERNEL_CASE(N) \
        case N: \
            hd_encoder_kernel<N><<<grid, threads, 0, stream>>>( \
                state->n_blk, \
                state->device.ngramm_sum_buffer, \
                state->device.item_lookup, \
                state->n_items, \
                d_x, n_x); \
            break;

        CALL_KERNEL_CASE(2)
        CALL_KERNEL_CASE(3)
        CALL_KERNEL_CASE(4)
        CALL_KERNEL_CASE(5)
        CALL_KERNEL_CASE(6)
        CALL_KERNEL_CASE(7)
        CALL_KERNEL_CASE(8)

        default:
            printf("Error! ngramm must be between 2 and 8, but it was %d\n", state->ngramm);
    }
}

extern "C" void hd_encoder_setup_device(struct hd_encoder_t * const state) {
    // allocate memory
    hipMalloc(&(state->device.item_lookup), state->n_items * state->n_blk * sizeof(block_t));
    hipMalloc(&(state->device.ngramm_sum_buffer), state->n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
    hipMalloc(&(state->device.ngramm_buffer), state->ngramm * state->n_blk * sizeof(block_t));

    // copy LUT to device
    hipMemcpy(
        state->device.item_lookup,
        state->item_lookup,
        state->n_items * state->n_blk * sizeof(block_t),
        hipMemcpyHostToDevice
    );
}

extern "C" void hd_encoder_free(struct hd_encoder_t * const state) {
    hipFree(state->device.item_lookup);
    hipFree(state->device.ngramm_sum_buffer);
    hipFree(state->device.ngramm_buffer);

    free(state->ngramm_buffer);
    free(state->ngramm_sum_buffer);
    free(state->item_buffer);
    free(state->item_lookup);

    hipDeviceReset();
}

extern "C" void hd_encoder_encode (
    struct hd_encoder_t * const state,
    const feature_t * const x,
    const int n_x
)
{
    const int n_blk = state->n_blk;

    // reset the sum count and buffer
    state->ngramm_sum_count = 0;
    hipMemset(
        state->device.ngramm_sum_buffer,
        0,
        n_blk * sizeof(block_t) * 8 * sizeof(uint32_t)
    );

    // allocate input data memory on the device
    feature_t * d_x;
    hipMalloc(&d_x, n_x * sizeof(feature_t));
    // copy the input data
    hipMemcpy(d_x, x, n_x * sizeof(feature_t), hipMemcpyHostToDevice);

    // call the kernel
    hd_encoder_call_kernel(state, d_x, n_x);

    hipDeviceSynchronize();

    // copy the output (ngramm_sum_buffer) back from the device
    hipMemcpy(
        state->ngramm_sum_buffer,
        state->device.ngramm_sum_buffer,
        n_blk * sizeof(block_t) * 8 * sizeof(uint32_t),
        hipMemcpyDeviceToHost
    );

    // free input memory
    hipFree(d_x);

    // set the ngramm_sum_count
    state->ngramm_sum_count += n_x - (state->ngramm - 1);
}

void clip(
    const uint32_t * const in,
    const int n_in,
    const int count,
    block_t * const out
)
{
    int threshold = count / 2;

    memset(out, 0, (n_in + sizeof(block_t) * 8 - 1) / (sizeof(block_t) * 8));

    // we ignore the randomization here...

    int n_blk = n_in / 32;
    int blk_idx;
    for (blk_idx = 0; blk_idx < n_blk; blk_idx++) {
        int i;
        for (i = 0; i < 32; i++) {
            out[blk_idx] <<= 1;
            out[blk_idx] += ((uint32_t)(threshold - in[i * n_blk + blk_idx])) >> 31;
        }
    }

}

void hd_encoder_clip(
    struct hd_encoder_t * const state
)
{
    clip(
        state->ngramm_sum_buffer,
        sizeof(block_t) * 8 * state->n_blk,
        state->ngramm_sum_count,
        state->ngramm_buffer
    );
}