#include "hip/hip_runtime.h"
#include <string.h>
#include <stdint.h>
#include <stdlib.h>

extern "C" {
#include "hd_batch_encoder.h"
}

__global__ void hd_encoder_kernel(
    const int n_blk,
    const int ngramm,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    const int n_items,
    const feature_t * __restrict__ x,
    const int n_x
);

__global__ void hd_encoder_3gramm_kernel(
    const int n_blk,
    uint32_t * __restrict__ ngramm_sum_buffer,
    const block_t * __restrict__ item_lookup,
    const int n_items,
    const feature_t * __restrict__ x,
    const int n_x
);

extern "C" void hd_batch_encoder_init(
     struct hd_encoder_t * const states,
     const int batch_size
)
{
    int n_blk = states[0].n_blk;
    int ngramm = states[0].ngramm;
    int n_items = states[0].n_items;

    int i;
    for (i = 1; i < batch_size; i++) {
        // initialize the state
        hd_encoder_init(&(states[i]), n_blk, ngramm, n_items);
        // free up the item lookup since we do use the same as the first one
        free(states[i].item_lookup);
        // use the same item lookup as first element in batch
        states[i].item_lookup = states[0].item_lookup;
    }
}

extern "C" void hd_batch_encoder_setup_device(
    struct hd_encoder_t * const states,
    const int batch_size
)
{
    int i;
    for (i = 0; i < batch_size; i++) {
        // initialize the state
        hipMalloc(&(states[i].device.ngramm_sum_buffer), states[i].n_blk * sizeof(block_t) * 8 * sizeof(uint32_t));
        hipMalloc(&(states[i].device.ngramm_buffer), states[i].ngramm * states[i].n_blk * sizeof(block_t));

        // only the first should copy the item lookup
        if (i == 0) {
            hipMalloc(&(states[i].device.item_lookup), states[i].n_items * states[i].n_blk * sizeof(block_t));
            hipMemcpy(
                states[i].device.item_lookup,
                states[i].item_lookup,
                states[i].n_items * states[i].n_blk * sizeof(block_t),
                hipMemcpyHostToDevice
            );
        } else {
            // use the same item lookup as first element in batch
            states[i].device.item_lookup = states[0].device.item_lookup;
        }
    }
}

void hd_batch_encoder_free(
    struct hd_encoder_t * const states,
    const int batch_size
)
{
    int i;
    for (i = 0; i < batch_size; i++) {
        hipFree(states[i].device.ngramm_sum_buffer);
        hipFree(states[i].device.ngramm_buffer);

        free(states[i].ngramm_buffer);
        free(states[i].ngramm_sum_buffer);
        free(states[i].item_buffer);
        if (i == 0) {
            free(states[i].item_lookup);
            hipFree(states[i].device.item_lookup);
        }
    }

    hipDeviceReset();
}

void hd_batch_encoder_encode (
    struct hd_encoder_t * states,
    const int batch_size,
    const feature_t ** const x,
    const int * const n_x
)
{
    // setup streams
    hipStream_t * streams = (hipStream_t*) malloc(sizeof(hipStream_t) * batch_size);
    int i;
    for (i = 0; i < batch_size; i++) {
        hipStreamCreate(&(streams[i]));
    }

    const int n_blk = states[0].n_blk;
    const int ngramm = states[0].ngramm;
    const int n_items = states[0].n_items;

    // array for device pointers
    feature_t ** d_x = (feature_t**) malloc(sizeof(feature_t *) * batch_size);

    // start every kernel
    for (i = 0; i < batch_size; i++) {
        // reset the sum count
        states[i].ngramm_sum_count = 0;

        // allocate input data memory on the device
        hipMalloc(&(d_x[i]), n_x[i] * sizeof(feature_t));
        // copy the input data
        hipMemcpyAsync(d_x[i], x[i], n_x[i] * sizeof(feature_t), hipMemcpyHostToDevice, streams[i]);

        // call the kernel
        int num_blocks = (n_blk + NUM_THREADS_IN_BLOCK - 1) / NUM_THREADS_IN_BLOCK;
        if (ngramm == 3) {
            hd_encoder_3gramm_kernel<<<num_blocks, NUM_THREADS_IN_BLOCK, 0, streams[i]>>>(
                n_blk,
                states[i].device.ngramm_sum_buffer,
                states[i].device.item_lookup,
                n_items,
                d_x[i],
                n_x[i]
            );
        } else {
            hd_encoder_kernel<<<num_blocks, NUM_THREADS_IN_BLOCK, 0, streams[i]>>>(
                n_blk,
                ngramm,
                states[i].device.ngramm_sum_buffer,
                states[i].device.item_lookup,
                n_items,
                d_x[i],
                n_x[i]
            );
        }

        // copy the output (ngramm_sum_buffer) back from the device
        hipMemcpyAsync(
            states[i].ngramm_sum_buffer,
            states[i].device.ngramm_sum_buffer,
            n_blk * sizeof(block_t) * 8 * sizeof(uint32_t),
            hipMemcpyDeviceToHost,
            streams[i]
        );

        // set the ngramm_sum_count
        states[i].ngramm_sum_count += n_x[i] - (ngramm - 1);
    }

    // wait until batch is complete
    hipDeviceSynchronize();

    // free up all the input data memory on the device
    for (i = 0; i < batch_size; i++) {
        hipFree(d_x[i]);
    }
    // free up the array holding all the device pointers
    free(d_x);
    // free up the streams
    free(streams);
}